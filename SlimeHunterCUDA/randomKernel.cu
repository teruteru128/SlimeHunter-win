#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include "jrandom.cuh"
#include "mcSlimeChunkOracle.cuh"

__host__ __device__ static int64_t initialScrambleKernel(uint64_t seed) {
	return (seed ^ MULTIPLIER) & MASK;
}

__host__ __device__ Random* setSeedKernel(Random *rnd, int64_t seed) {
	rnd->seed = initialScrambleKernel(seed);
	return rnd;
}

__host__ __device__ int32_t nextKernel(Random* rnd, int32_t bits) {
	uint64_t oldseed, nextseed;
	uint64_t seed = rnd->seed;
	oldseed = seed;
	nextseed = (oldseed * MULTIPLIER + ADDEND) & MASK;
	rnd->seed = nextseed;
	return (uint32_t)(nextseed >> (48 - bits));
}

__host__ __device__ int64_t nextLongKernel(Random* rnd) {
	return ((int64_t)nextKernel(rnd, 32) << 32) + nextKernel(rnd, 32);
}

__host__ __device__ int32_t nextIntKernel(Random* rnd) {
	return nextKernel(rnd, 32);
}

__host__ __device__ int32_t nextIntWithRangeKernel(Random *rnd, int32_t bound) {
	if (bound <= 0) {
		// err!
		return 0;
	}
	int32_t r = nextKernel(rnd, 31);
	int32_t m = bound - 1;
	if ((bound & m) == 0) {
		r = (int32_t)((bound * (int64_t)r) >> 31);
	}
	else {
		int32_t u;
		for (u = r; u - (r = u % bound) + m < 0; u = nextKernel(rnd, 31));
	}
	return r;
}

#if 1
__device__ SlimeChunkSeed* setMCSeedKernel(SlimeChunkSeed* t, int64_t seed) {
	t->seed = seed;
	return t;
}

__device__ int64_t getMCSeedKernel(SlimeChunkSeed* seed, int32_t chunkX, int32_t chunkZ) {
	return seed->seed + chunkX * chunkX * 0x4c1906 + chunkX * 0x5ac0db + chunkZ * chunkZ * 0x4307a7L + chunkZ * 0x5f24f ^ 0x3ad8025f;
}

__device__ static bool isSlimeChunkKernel(Random* rnd) {
	return nextIntWithRangeKernel(rnd, 10) == 0;
}

__device__ bool isSlimeChunkXZKernel(SlimeChunkSeed* seed, int64_t chunkX, int64_t chunkZ) {
	return isSlimeChunkKernel(setSeedKernel(&seed->rnd, getMCSeedKernel(seed, (int32_t)chunkX, (int32_t)chunkZ)));
}
#else

__device__ SlimeChunkSeed* setMCSeedKernel(SlimeChunkSeed* t, int64_t seed) {
	t->seed = seed;
	return t;
}

__device__ int64_t getMCSeedKernel(SlimeChunkSeed* seed, int32_t chunkX, int32_t chunkZ) {
	return seed->seed + chunkX * chunkX * 0x4c1906 + chunkX * 0x5ac0db + chunkZ * chunkZ * 0x4307a7L + chunkZ * 0x5f24f ^ 0x3ad8025f;
}

__device__ bool isSlimeChunkKernel(Random* rnd) {
	//return nextIntWithRangeKernel(rnd, 10) == 0;
	return false;
}

__device__ bool isSlimeChunkXZKernel(SlimeChunkSeed* mCSeed, int64_t chunkX, int64_t chunkZ) {
	//int64_t seed = getMCSeedKernel(mCSeed, (int32_t)chunkX, (int32_t)chunkZ);
	//Random *rnd = setSeedKernel(&mCSeed->rnd, seed);
	//return isSlimeChunkKernel(rnd);
	isSlimeChunkKernel(&mCSeed->rnd);
	return 0;
}
#endif

__global__ void genRandomKernel(int64_t *out, const int64_t *in, Random *random, const size_t n) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n) {
		return;
	}
	out[i] = nextLongKernel(setSeedKernel(&random[i], in[i]));
	//out[i] = in[i];
}

hipError_t randomWithCuda(int64_t * out, int64_t *in, Random* random, size_t n) {
	int64_t *dev_in = NULL;
	int64_t *dev_out = NULL;
	Random *dev_random = NULL;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed\n");
		goto error;
	}

	cudaStatus = hipMalloc(&dev_out, n * sizeof(int64_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed(dev_out)\n");
		goto error;
	}

	cudaStatus = hipMalloc(&dev_in, n * sizeof(int64_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed(dev_in)\n");
		goto error;
	}

	cudaStatus = hipMalloc(&dev_random, n * sizeof(Random));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed(dev_in)\n");
		goto error;
	}

	cudaStatus = hipMemcpy(dev_in, in, n * sizeof(int64_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed(dev_in)\n");
		goto error;
	}

	cudaStatus = hipMemcpy(dev_random, random, n * sizeof(Random), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed(dev_in)\n");
		goto error;
	}

	genRandomKernel << <1, n >> > (dev_out, dev_in, dev_random, n);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(out, dev_out, n * sizeof(int64_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto error;
	}
error:
	hipFree(dev_in);
	hipFree(dev_out);
	hipFree(dev_random);
	return cudaStatus;
}
