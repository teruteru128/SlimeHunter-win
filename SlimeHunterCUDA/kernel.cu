
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include <windows.h>
#include <time.h>
#include <random>
#include "jrandom.cuh"
#include "add.cuh"
#include "randomKernel.cuh"


int main(int argc, char* argv[])
{
	int arraySize = 5;
	const int a[5] = {
		1, 2, 3, 4, 5
	};
	const int b[5] = {
		10, 20, 30, 40, 50
	};
	int c[5] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	printf("{1, 2, 3, 4, 5} +{ 10, 20, 30, 40, 50, 60, 70, 80, 90, 100} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);
	
	size_t n = 334;
	int64_t *in = NULL;
	int64_t *out = NULL;
	Random *random = NULL;

	in = (int64_t *)calloc(n, sizeof(size_t));
	out = (int64_t *)calloc(n, sizeof(size_t));
	random = (Random *)calloc(n, sizeof(Random));

	std::random_device rnd;
	std::mt19937 mt(rnd());

	for (int i = 0; i < n; i++) {
		in[i] = ((int64_t)mt()) <<32 + mt();
	}

	cudaStatus = randomWithCuda(out, in, random, n);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "randomWithCuda failed!");
		return 1;
	}
	int i;
	
	for (i = 0; i < n; i++) {
		printf("%5s,%" PRIu64",%" PRIu64"\n", out[i] == in[i] ? "true": "false",out[i], in[i]);
	}
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}
